#include <stdio.h>
#include <string.h>
#include <math.h>

// includes cuda libraries
#include <cutil.h>

// includes kernel
#include <template_kernel.cu>
//#include <CNU_kernel.cu>
//#include <VNU_kernel.cu>



void ldpc( int argc, char** argv);                                  //Function called from main


//All of the following functions are located in .cpp file
extern "C" 
void computeNumSeq(int*, char*);
extern "C" 
void create_Host_Array(char*, short int*, int);
extern "C" 
void write_Results(char*, char*, int*);
extern "C" 
void create_Offset_Array(short int*, int);


/*______________________________________________Program Main___________________________________________________*/


int main( int argc, char** argv)
{
    ldpc( argc, argv);

    CUT_EXIT(argc, argv);
}




/*______________________________________________ldpc function_________________________________________________*/

void
ldpc( int argc, char** argv)
{

//Searches for first available CUDA device
    CUT_DEVICE_INIT(argc, argv);



/*________________________________Allocate memory on host to hold file name______________________________________*/


//Allocate memory for the files location/path on PC limited to 200 characters
char* database_Path = (char*) malloc(sizeof(char) * 200);





/*________________________________Allocate memory on host for integer Array______________________________________*/

//integer array on host. use short ints to save memory
short int* host_array = (short int*) malloc(sizeof(short int) * 100);




/*________________________________Allocate memory on Device for integer Array____________________________________*/

//integer array on device. use short ints to save memory
short int* device_array;
short int* device_offset;
CUDA_SAFE_CALL(hipMalloc((void**) &device_array, (100 * sizeof(short int)) ));





/*______________________________________Create Array on host from File___________________________________________*/




/*_______________________________________Copy Host Array Contents to Device Array_________________________________*/





/*________________________________Allocate memory on host for offset Array______________________________________*/




/*________________________________Allocate memory on Device for offset Array____________________________________*/



/*______________________________________Create Array on host___________________________________________*/






/*_______________________________________Copy Host Array Contents to Device Array_________________________________*/






/*___________________________Start timer to measure kernel execution time______________________________________*/


//Start Timer
unsigned int timer = 0;                                //Create a variable timer and set it to zero
CUT_SAFE_CALL( cutCreateTimer( &timer));               //Creates a timer and sends result to variable timer
CUT_SAFE_CALL( cutStartTimer( timer));                 //Starts the execution of the timer

printf( "\nLaunching Kernel... \n");




/*_________________________________Setup Execution Parameters For CNU Kernel____________________________________*/


//Number of Threads per block
int numThreads = 256;
int numBlocks = 48;

// setup execution parameters
dim3 threads(numThreads);
dim3 grid(numBlocks);


/*_____________________________________________Execute CNU Kernel__________________________________________________*/


// execute the kernel
//CNU_kernel<<< grid, threads>>>(device_array, device_offset);
template_kernel<<< grid, threads>>>(device_array, device_offset);



/*_________________________________Setup Execution Parameters For VNU Kernel____________________________________*/





/*_____________________________________________Execute VNU Kernel__________________________________________________*/





/*_______________________________________Copy Results from GPU to Host_________________________________________*/




/*_______________________________________________Check Results_________________________________________________*/







/*______________________________________________Clean Up Data__________________________________________________*/


    //Free Device Memory
    CUDA_SAFE_CALL(hipFree(device_array));
}