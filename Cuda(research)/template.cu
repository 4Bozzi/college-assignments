#include <stdio.h>
#include <string.h>
#include <math.h>

// includes cuda libraries
#include <cutil.h>
#include <iostream>


// includes kernel
#include <template_kernel.cu>
//#include <CNU_kernel.cu>
//#include <VNU_kernel.cu>



void ldpc( int argc, char** argv);                                  //Function called from main


//All of the following functions are located in .cpp file
extern "C" 
void compute_Number_Of_Integers(int*);
extern "C" 
void create_Host_Array(short int*);
extern "C" 
void write_Results(short int*, char*, int*);
extern "C" 
void create_Offset_Array(short int* offsetArray, int* number_of_integers);
extern "C" 
void sequential_CNVN(short int*, int*, short int*, short int*);
extern "C" 
void create_Results_Array(short int* results_array, int* number_of_integers);
extern "C" 
void create_VNSign_Array(short int* signArray, int* number_of_integers);
extern "C" 
int calculate_Sign(short int* sign_array, int* number_of_integers, short int* signcalc_array );


/*______________________________________________Program Main___________________________________________________*/
int main( int argc, char** argv)
{
    ldpc( argc, argv);

    CUT_EXIT(argc, argv);
}




/*______________________________________________ldpc function_________________________________________________*/
void
ldpc( int argc, char** argv)
{

//Searches for first available CUDA device
    CUT_DEVICE_INIT(argc, argv);

int sign = 0;


/*______________________________________________compute number_of_integers_________________________________________________*/
int number_of_integers = 0;
compute_Number_Of_Integers(&number_of_integers);
std::cout << "Number of integers: " << number_of_integers << std::endl;




/*________________________________Allocate memory on host to hold file name______________________________________*/
//Allocate memory for the files location/path on PC limited to 200 characters
char* database_Path = (char*) malloc(sizeof(char) * 200);




/*________________________________Allocate memory on host for RESULTS Array______________________________________*/
//integer array on host. use short ints to save memory
short int* host_results_array = (short int*) malloc(sizeof(short int) * number_of_integers * 2);




/*________________________________Create RESULTS Array______________________________________*/
//sets all of the values to 0 initialy
create_Results_Array(host_results_array, &number_of_integers);

////////////////////////////////////////////////////////////////////////////////////////////////
//Testing to make sure that results array now contains all zeros
//	for(int cnt = 0; cnt < number_of_integers; cnt++){
//		printf("resultsArray[%d] = %d\n", cnt, results_array[cnt]);
//	}
////////////////////////////////////////////////////////////////////////////////////////////////


/*________________________________Allocate memory on device for RESULTS Array______________________________________*/
short int* results_array = (short int*) malloc(sizeof(short int) * number_of_integers * 2);
CUDA_SAFE_CALL(hipMalloc((void**) &results_array, ((number_of_integers * sizeof(short int))*2) ));




/*________________________________Allocate memory on host for SIGN Array______________________________________*/
//integer array on host use short ints to save memory
short int* sign_array = (short int*) malloc(sizeof(short int) * number_of_integers);




/*__________________________________________Create SIGN Array__________________________________________________*/
//sets all of the values to 0 initialy
create_VNSign_Array(sign_array, &number_of_integers);

////////////////////////////////////////////////////////////////////////////////////////////////
//Testing to make sure that sign array now contains all zeros
//	for(int cnt = 0; cnt < number_of_integers; cnt++){
//		printf("signArray[%d] = %d\n", cnt, sign_array[cnt]);
//	}
////////////////////////////////////////////////////////////////////////////////////////////////



/*________________________________Allocate memory on device for SIGN Array______________________________________*/
short int* device_sign_array = (short int*) malloc(sizeof(short int) * number_of_integers);
CUDA_SAFE_CALL(hipMalloc((void**) &device_sign_array, (number_of_integers * sizeof(short int)) ));





/*__________________________Allocate memory on host for SIGN CALULATIONS Array__________________________________*/
short int* signcalc_array = (short int*) malloc(sizeof(short int) * (number_of_integers/3));




/*________________________________Allocate memory on host for OFFSET Array______________________________________*/

short int* host_offset_array = (short int*) malloc(sizeof(short int) * number_of_integers);




/*__________________________________________Create OFFSET Array__________________________________________________*/
//sets all of the values to 0 initialy
create_Offset_Array(host_offset_array, &number_of_integers);

////////////////////////////////////////////////////////////////////////////////////////////////
//Testing to make sure that offset array now contains all zeros
//	for(int cnt = 0; cnt < number_of_integers; cnt++){
//		printf("offsetArray[%d] = %d\n", cnt, offset_array[cnt]);
//	}
////////////////////////////////////////////////////////////////////////////////////////////////



/*________________________________Allocate memory on Device for OFFSET Array______________________________________*/

//offset array on device. use short ints to save memory
short int* offset_array = (short int*) malloc(sizeof(short int) * number_of_integers);
CUDA_SAFE_CALL(hipMalloc((void**) &offset_array, (number_of_integers * sizeof(short int)) ));




/*________________________________Allocate memory on host for INTEGER Array______________________________________*/

//integer array on host. use short ints to save memory
short int* host_array = (short int*) malloc(sizeof(short int) * number_of_integers);



/*________________________________Allocate memory on Device for INTEGER  Array____________________________________*/

//integer array on device. use short ints to save memory
short int* device_array = (short int*) malloc(sizeof(short int) * number_of_integers);
CUDA_SAFE_CALL(hipMalloc((void**) &device_array, (number_of_integers * sizeof(short int)) ));





/*____________________________________Create INTEGER Array on host from File________________________________________*/
//Read integers into array located on the PC using a function which you will create in the .cpp file
create_Host_Array(host_array);

////////////////////////////////////////////////////////////////////////////////////////////////
//Testing to make sure that integer array now contains all zeros
//	for(int cnt = 0; cnt < number_of_integers; cnt++){
//		printf("hostArray[%d] = %d\n", cnt, host_array[cnt]);
//	}
////////////////////////////////////////////////////////////////////////////////////////////////



/*_______________________________________Copy Host Array Contents to Device Array_________________________________*/
// copy host memory database array to allocated device array
CUDA_SAFE_CALL(hipMemcpy(device_array, host_array, (number_of_integers * sizeof(short int)), hipMemcpyHostToDevice));




/*_________________________________Setup Execution Parameters For VNU Kernel____________________________________*/
//Number of Threads per block
short int numThreads = 64;
short int numBlocks = 96;

// setup execution parameters
dim3 threads(numThreads);
dim3 grid(numBlocks);

/*___________________________Start timer to measure kernel execution time______________________________________*/
printf( "\nLaunching Multithreaded Kernel... \n");
//Start Timer
unsigned int timer = 0;                                //Create a variable timer and set it to zero
CUT_SAFE_CALL( cutCreateTimer( &timer));               //Creates a timer and sends result to variable timer
CUT_SAFE_CALL( cutStartTimer( timer));                 //Starts the execution of the timer




/*__________________________________________Execute VNU Kernel_________________________________________________*/
// execute the vn kernel
VNU_kernel<<<grid, threads>>>(device_array, offset_array, device_sign_array, results_array);




/*_________________________________Setup Execution Parameters For CNU Kernel____________________________________*/
//Number of Threads per block
//short int numThreads = 32;
//short int numBlocks = 3;

// setup execution parameters
//dim3 threads(numThreads);
//dim3 grid(numBlocks);


/*_________________________________________Execute CNU Kernel___________________________________________________*/
CNU_kernel<<<grid, threads>>>(offset_array, results_array);




/*_____________________________________________Check Results____________________________________________________*/
//Stop Timer
CUT_SAFE_CALL( cutStopTimer( timer));
printf( "\nGPU database scan time: %f (ms)\n", cutGetTimerValue( timer));
CUT_SAFE_CALL( cutDeleteTimer( timer));




/*_______________________________________Copy Results from GPU to Host__________________________________________*/
printf( "\nCopying Results from GPU to host... \n");

//Copy Results from GPU
CUDA_SAFE_CALL(hipMemcpy(host_array, results_array, (number_of_integers * sizeof(short int)), hipMemcpyDeviceToHost));
CUDA_SAFE_CALL(hipMemcpy(sign_array, device_sign_array, (number_of_integers * sizeof(short int)), hipMemcpyDeviceToHost));



/*_____________________________________Sign Array Calculations___________________________________________________*/
sign = calculate_Sign(sign_array, &number_of_integers, signcalc_array );


printf( "\nThe value of the sign multithreaded operations is: %d\n", sign);



/*_______________________________Copy Results from Host to Global Memory_______________________________________*/
char path[20];
write_Results(host_array, path, &number_of_integers);








/*---------------------------------Sequential Version for Comparison-------------------------------------------*/

//resets host arrays to original values
create_Host_Array(host_array);
create_Offset_Array(host_offset_array, &number_of_integers);
create_VNSign_Array(sign_array, &number_of_integers);
create_Results_Array(host_results_array, &number_of_integers);


/*___________________________Start timer to measure kernel execution time______________________________________*/

printf( "\nLaunching Sequential Kernel... \n");
//Start Timer
timer = 0;										       //Create a variable timer and set it to zero
CUT_SAFE_CALL( cutCreateTimer( &timer));               //Creates a timer and sends result to variable timer
CUT_SAFE_CALL( cutStartTimer( timer));                 //Starts the execution of the timer


//make a call to the sequential version
sequential_CNVN(host_array, &number_of_integers, sign_array, host_results_array);



/*_______________________________________________Check Results_________________________________________________*/

//Stop Timer
CUT_SAFE_CALL( cutStopTimer( timer));
printf( "\nCPU database scan time: %f (ms)\n", cutGetTimerValue( timer));
CUT_SAFE_CALL( cutDeleteTimer( timer));

sign = calculate_Sign(sign_array, &number_of_integers, signcalc_array);
printf( "\nThe value of the sign sequential operations is: %d\n", sign);

/*_______________________________Copy Results from Host to Global Memory_______________________________________*/
write_Results(host_results_array, path, &number_of_integers);


/*----------------------------------------End Sequencial Version-----------------------------------------------*/









/*______________________________________________Clean Up Data__________________________________________________*/

    //Free Device Memory
    CUDA_SAFE_CALL(hipFree(device_array));
}