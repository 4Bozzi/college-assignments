#include <stdio.h>
#include <string.h>
#include <math.h>

// includes cuda libraries
#include <cutil.h>

// includes kernel
#include <template_kernel.cu>
//#include <CNU_kernel.cu>
//#include <VNU_kernel.cu>



void ldpc( int argc, char** argv);                                  //Function called from main


//All of the following functions are located in .cpp file
extern "C" 
void computeNumSeq(int*, char*);
extern "C" 
void create_Host_Array(char*, short int*, int);
extern "C" 
void write_Results(char*, char*, int*);
extern "C" 
void create_Offset_Array(short int*, int);


/*______________________________________________Program Main___________________________________________________*/


int main( int argc, char** argv)
{
    ldpc( argc, argv);

    CUT_EXIT(argc, argv);
}




/*______________________________________________ldpc function_________________________________________________*/

void
ldpc( int argc, char** argv)
{

//Searches for first available CUDA device
    CUT_DEVICE_INIT(argc, argv);



/*________________________________Allocate memory on host to hold file name______________________________________*/


//Allocate memory for the files location/path on PC limited to 200 characters
char* database_Path = (char*) malloc(sizeof(char) * 200);





/*________________________________Allocate memory on host for integer Array______________________________________*/

//integer array on host. use short ints to save memory
short int* host_array = (short int*) malloc(sizeof(short int) * 100);




/*________________________________Allocate memory on Device for integer Array____________________________________*/

//integer array on device. use short ints to save memory
short int* device_array[];
short int* device_offset = 0;
CUDA_SAFE_CALL(hipMalloc((void**) &device_array, (100 * sizeof(short int)) ));





/*______________________________________Create Array on host from File___________________________________________*/


//Read integers into array located on the PC using a function which you will create in the .cpp file
create_Host_Array(database_Path, host_array, 100);


//Print values of host array now
//(for i < number_of_integers)
//printf(%d, host_array[i])



/*_______________________________________Copy Host Array Contents to Device Array_________________________________*/


// copy host memory database array to allocated device array
CUDA_SAFE_CALL(hipMemcpy(device_array, host_array, (100 * sizeof(short int)), hipMemcpyHostToDevice));



//not sure if i did the offset array correctly


/*________________________________Allocate memory on host for offset Array______________________________________*/


//offset array on host. use short ints to save memory
short int* host_offset_array = (short int*) malloc(sizeof(short int) * 100);




/*________________________________Allocate memory on Device for offset Array____________________________________*/

//offset array on device. use short ints to save memory
CUDA_SAFE_CALL(hipMalloc((void**) &device_offset, (100 * sizeof(short int)) ));





/*______________________________________Create Array on host___________________________________________*/


//Read integers into array located on the PC using a function which you will create in the .cpp file
create_Offset_Array(host_offset_array, 100);






/*_______________________________________Copy Host Array Contents to Device Array_________________________________*/


// copy host memory database array to allocated device array
CUDA_SAFE_CALL(hipMemcpy(device_array, host_array, (100 * sizeof(short int)), hipMemcpyHostToDevice));







/*___________________________Start timer to measure kernel execution time______________________________________*/


//Start Timer
unsigned int timer = 0;                                //Create a variable timer and set it to zero
CUT_SAFE_CALL( cutCreateTimer( &timer));               //Creates a timer and sends result to variable timer
CUT_SAFE_CALL( cutStartTimer( timer));                 //Starts the execution of the timer

printf( "\nLaunching Kernel... \n");




/*_________________________________Setup Execution Parameters For CNU Kernel____________________________________*/


//Number of Threads per block
int numThreads = 256;
int numBlocks = 48;

// setup execution parameters
dim3 threads(numThreads);
dim3 grid(numBlocks);


/*_____________________________________________Execute CNU Kernel__________________________________________________*/


// execute the kernel
//CNU_kernel<<< grid, threads>>>(device_array, device_offset);
template_kernel<<< grid, threads>>>(device_array, device_offset);

// check if kernel execution generated an error
CUT_CHECK_ERROR("Kernel execution failed");                        //Report error if kernel did not launch



/*_________________________________Setup Execution Parameters For VNU Kernel____________________________________*/


//Number of Threads per block
//int numThreads = 256;
//int numBlocks = 48;

// setup execution parameters
//dim3 threads(numThreads);
//dim3 grid(numBlocks);


/*_____________________________________________Execute VNU Kernel__________________________________________________*/


// execute the kernel
//VNU_kernel<<< grid, threads>>>(device_array);

// check if kernel execution generated an error
//CUT_CHECK_ERROR("Kernel execution failed");                        //Report error if kernel did not launch




/*_______________________________________Copy Results from GPU to Host_________________________________________*/

printf( "\nCopying Results from GPU to host... \n");

//Copy Results from GPU
CUDA_SAFE_CALL(hipMemcpy(host_array, device_array, (100 * sizeof(short int)), hipMemcpyDeviceToHost));





/*_______________________________________________Check Results_________________________________________________*/


//Stop Timer
CUT_SAFE_CALL( cutStopTimer( timer));
printf( "\nGPU database scan time: %f (ms)\n", cutGetTimerValue( timer));
CUT_SAFE_CALL( cutDeleteTimer( timer));





/*______________________________________________Clean Up Data__________________________________________________*/
    //Free Host Memory
    free(host_array);

    //Free Device Memory
    CUDA_SAFE_CALL(hipFree(device_array));
}